#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdint>

#include "RayTracer.cuh"
#include "../DrawValues.h"
#include "../../scene/structs/Scene.h"
#include "../../scene/structs/SceneObject.h"
#include "../../math/cuda/CudaMath.cuh"

#include "kernels/RaytraceKernel.cuh"

#include "../Profiling.h"

void cudaAllocateScene()
{
  sceneStructSize = sizeof(Scene);

  // Allocate all the memory needed - * means cudamemcpy'd each frame
  hipMalloc(&scene->d_pointarray, scene->pointsSize);
  hipMalloc(&scene->d_trsfrmdpoints, scene->pointsSize);
  hipMalloc(&scene->d_pointToObjIdxTable, scene->pointTableSize);
  hipMalloc(&scene->d_triangles, scene->triangleSize);
  hipMalloc(&scene->d_sceneobjects, scene->sceneObjectsSize);
  hipMalloc(&d_scene, sceneStructSize); // *

  // Initial cudamemcpy
  hipMemcpy(scene->d_pointarray, scene->points, scene->pointsSize, hipMemcpyHostToDevice);
  hipMemcpy(scene->d_triangles, scene->triangles, scene->triangleSize, hipMemcpyHostToDevice);
  hipMemcpy(scene->d_pointToObjIdxTable, scene->pointToObjIdxTable, scene->pointTableSize, hipMemcpyHostToDevice);
  hipMemcpy(scene->d_sceneobjects, scene->sceneobjects, scene->sceneObjectsSize, hipMemcpyHostToDevice);
  hipMemcpy(d_scene, scene, sceneStructSize, hipMemcpyHostToDevice);
}

void cudaCleanup()
{
  hipHostFree(scene->transformMatrices);

  hipFree(scene->d_pointarray);
  hipFree(scene->d_trsfrmdpoints);
  hipFree(scene->d_triangles);
  hipFree(scene->d_sceneobjects);
  hipFree(scene->d_transformMatrices);
  hipFree(d_scene);

  hipHostFree(scene);
}

void rayTrace(
    uchar4 *pixelBuffer,
    const int bgColor)
{
  ZONESCOPEDNC("rayTrace function", PROFILER_LIME_GREEN);
  TRACYCZONENC(cudaTrace, "Cuda trace", true, PROFILER_GOLD);

  constexpr dim3 blockDim(RAYTRACE_BLOCK_SIDE, RAYTRACE_BLOCK_SIDE);
  constexpr dim3 gridDim((WIDTH + (RAYTRACE_BLOCK_SIDE - 1)) / RAYTRACE_BLOCK_SIDE,
                         (HEIGHT + (RAYTRACE_BLOCK_SIDE - 1)) / RAYTRACE_BLOCK_SIDE);

  float3_L f3lBg = intColToF3l(bgColor);

  rayTraceKernel<<<gridDim, blockDim>>>(
      pixelBuffer,
      d_scene,
      WIDTH, HEIGHT,
      f3lBg);

  if (scene->afterTraceSync)
    hipDeviceSynchronize();

  TRACYCZONEEND(cudaTrace);
}