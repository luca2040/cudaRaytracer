#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdint>

#include "RayTracer.cuh"
#include "../DrawValues.h"
#include "../../scene/structs/Scene.h"
#include "../../scene/structs/SceneObject.h"
#include "../../math/cuda/CudaMath.cuh"

#include "kernels/RaytraceKernel.cuh"

#include "../Profiling.h"

void cudaAllocateScene()
{
  sceneStructSize = sizeof(Scene);

  // Allocate all the memory needed - * means cudamemcpy'd each frame
  hipMalloc(&scene->d_pointarray, scene->pointsSize);
  hipMalloc(&scene->d_trsfrmdpoints, scene->pointsSize);
  hipMalloc(&scene->d_pointToObjIdxTable, scene->pointTableSize);
  hipMalloc(&scene->d_triangles, scene->triangleSize);
  hipMalloc(&scene->d_sceneobjects, scene->sceneObjectsSize);
  hipMalloc(&d_scene, sceneStructSize); // *

  // Initial cudamemcpy
  hipMemcpy(scene->d_pointarray, scene->points, scene->pointsSize, hipMemcpyHostToDevice);
  hipMemcpy(scene->d_triangles, scene->triangles, scene->triangleSize, hipMemcpyHostToDevice);
  hipMemcpy(scene->d_pointToObjIdxTable, scene->pointToObjIdxTable, scene->pointTableSize, hipMemcpyHostToDevice);
  hipMemcpy(scene->d_sceneobjects, scene->sceneobjects, scene->sceneObjectsSize, hipMemcpyHostToDevice);
  hipMemcpy(d_scene, scene, sceneStructSize, hipMemcpyHostToDevice);
}

void cudaCleanup()
{
  hipHostFree(scene->transformMatrices);

  hipFree(scene->d_pointarray);
  hipFree(scene->d_trsfrmdpoints);
  hipFree(scene->d_triangles);
  hipFree(scene->d_sceneobjects);
  hipFree(scene->d_transformMatrices);
  hipFree(d_scene);

  hipHostFree(scene);
}

void rayTrace(uchar4 *pixelBuffer, int renderWidth, int renderHeight)
{
  ZONESCOPEDNC("rayTrace function", PROFILER_LIME_GREEN);
  TRACYCZONENC(cudaTrace, "Cuda trace", true, PROFILER_GOLD);

  dim3 blockDim(RAYTRACE_BLOCK_SIDE, RAYTRACE_BLOCK_SIDE);
  dim3 gridDim((renderWidth + (RAYTRACE_BLOCK_SIDE - 1)) / RAYTRACE_BLOCK_SIDE,
               (renderHeight + (RAYTRACE_BLOCK_SIDE - 1)) / RAYTRACE_BLOCK_SIDE);

  rayTraceKernel<<<gridDim, blockDim>>>(
      pixelBuffer,
      d_scene,
      renderWidth, renderHeight);

  if (scene->afterTraceSync)
    hipDeviceSynchronize();

  TRACYCZONEEND(cudaTrace);
}