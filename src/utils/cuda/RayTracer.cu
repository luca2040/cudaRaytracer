#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdint>

#include "RayTracer.cuh"
#include "../DrawValues.h"
#include "../../scene/structs/Scene.h"
#include "../../scene/structs/SceneObject.h"
#include "../../math/cuda/CudaMath.cuh"

#include "kernels/RaytraceKernel.cuh"
#include "definitions/RenderingStructs.cuh"

#include "../../third_party/tracy/tracy/Tracy.hpp"

// ######## Device parameters ########

float3_L *d_pointarray;
triangleidx *d_triangles;
SceneObject *d_sceneobjects;

void cudaAllocateAndCopy(size_t pointsSize,
                         size_t triangleSize,
                         size_t sceneobjectsSize)
{
  // Allocate all the memory needed
  hipMalloc(&d_pointarray, pointsSize);
  hipMalloc(&d_triangles, triangleSize);
  hipMalloc(&d_sceneobjects, sceneobjectsSize);

  // Copy the triangles index array, since its always static - not now because normals need to be recalculated each frame
  // hipMemcpy(d_triangles, triangles, triangleSize, hipMemcpyHostToDevice);
}

void cudaCleanup()
{
  hipFree(d_pointarray);
  hipFree(d_triangles);
  hipFree(d_sceneobjects);
}

void rayTrace(
    uchar4 *pixelBuffer,
    const int bgColor)
{
  ZoneScopedN("rayTrace function");

  // hipMemcpy(d_pixelBuffer, pixelBuffer, pixelBufferSize, hipMemcpyHostToDevice);
  hipMemcpy(d_pointarray, scene.transformedPoints, scene.pointsSize, hipMemcpyHostToDevice);
  hipMemcpy(d_triangles, scene.triangles, scene.triangleSize, hipMemcpyHostToDevice);
  hipMemcpy(d_sceneobjects, scene.sceneobjects, scene.sceneObjectsSize, hipMemcpyHostToDevice);

  constexpr dim3 blockDim(16, 16);
  constexpr dim3 gridDim((WIDTH + 15) / 16, (HEIGHT + 15) / 16);

  float3_L f3lBg = intColToF3l(bgColor);
  SceneMemoryPointers memPointers = SceneMemoryPointers(d_pointarray, d_triangles, d_sceneobjects,
                                                        scene.sceneobjectsNum);

  auto &cam = scene.cam;

  rayTraceKernel<<<gridDim, blockDim>>>(
      pixelBuffer,
      cam.camPos, cam.camViewOrigin, cam.imageX, cam.imageY,
      cam.inverseWidthMinus, cam.inverseHeightMinus,
      memPointers,
      WIDTH, HEIGHT,
      f3lBg);
}