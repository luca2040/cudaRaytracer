#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdint>

#include "RayTracer.cuh"
#include "../DrawValues.h"
#include "../../scene/structs/SceneObject.h"
#include "../../math/cuda/CudaMath.cuh"

#include "kernels/RaytraceKernel.cuh"
#include "definitions/RenderingStructs.cuh"

#include "../../third_party/tracy/tracy/Tracy.hpp"

// ######## Device parameters ########

float3_L *d_pointarray;
triangleidx *d_triangles;
SceneObject *d_sceneobjects;

void cudaAllocateAndCopy(size_t pointsSize,
                         size_t triangleSize,
                         size_t sceneobjectsSize)
{
  // Allocate all the memory needed
  hipMalloc(&d_pointarray, pointsSize);
  hipMalloc(&d_triangles, triangleSize);
  hipMalloc(&d_sceneobjects, sceneobjectsSize);

  // Copy the triangles index array, since its always static - not now because normals need to be recalculated each frame
  // hipMemcpy(d_triangles, triangles, triangleSize, hipMemcpyHostToDevice);
}

void cudaCleanup()
{
  hipFree(d_pointarray);
  hipFree(d_triangles);
  hipFree(d_sceneobjects);
}

void rayTrace(
    uchar4 *pixelBuffer,

    const float3_L camPos,
    const float3_L camViewOrigin,
    const float3_L imageX,
    const float3_L imageY,
    float inverseWidthMinus,
    float inverseHeightMinus,

    const float3_L *pointarray,
    const triangleidx *triangles,
    const SceneObject *sceneobjects,
    size_t sceneobjectsNum,

    size_t pointarraySize,
    size_t trianglesSize,
    size_t sceneobjectsSize,

    const int bgColor)
{
  ZoneScopedN("rayTrace function");

  // hipMemcpy(d_pixelBuffer, pixelBuffer, pixelBufferSize, hipMemcpyHostToDevice);
  hipMemcpy(d_pointarray, pointarray, pointarraySize, hipMemcpyHostToDevice);
  hipMemcpy(d_triangles, triangles, trianglesSize, hipMemcpyHostToDevice);
  hipMemcpy(d_sceneobjects, sceneobjects, sceneobjectsSize, hipMemcpyHostToDevice);

  constexpr dim3 blockDim(16, 16);
  constexpr dim3 gridDim((WIDTH + 15) / 16, (HEIGHT + 15) / 16);

  float3_L f3lBg = intColToF3l(bgColor);
  SceneMemoryPointers memPointers = SceneMemoryPointers(d_pointarray, d_triangles, d_sceneobjects,
                                                        sceneobjectsNum);

  rayTraceKernel<<<gridDim, blockDim>>>(
      pixelBuffer,
      camPos, camViewOrigin, imageX, imageY,
      inverseWidthMinus, inverseHeightMinus,
      memPointers,
      WIDTH, HEIGHT,
      f3lBg);
}