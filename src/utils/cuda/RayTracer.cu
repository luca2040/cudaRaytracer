#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdint>

#include "RayTracer.cuh"
#include "../DrawValues.h"
#include "../../scene/structs/Scene.h"
#include "../../scene/structs/SceneObject.h"
#include "../../math/cuda/CudaMath.cuh"

#include "kernels/RaytraceKernel.cuh"

#include "../../third_party/tracy/tracy/Tracy.hpp"
#include "../../third_party/tracy/tracy/TracyC.h"

void cudaAllocateScene()
{
  sceneStructSize = sizeof(Scene);

  // Allocate all the memory needed - * means cudamemcpy'd each frame
  hipMalloc(&scene->d_pointarray, scene->pointsSize);
  hipMalloc(&scene->d_trsfrmdpoints, scene->pointsSize);
  hipMalloc(&scene->d_pointToObjIdxTable, scene->pointTableSize);
  hipMalloc(&scene->d_triangles, scene->triangleSize);
  hipMalloc(&scene->d_sceneobjects, scene->sceneObjectsSize);
  hipMalloc(&scene->d_transformMatrices, scene->matricesSize); // *
  hipMalloc(&d_scene, sceneStructSize);                        // *

  // Initial cudamemcpy
  hipMemcpy(scene->d_pointarray, scene->points, scene->pointsSize, hipMemcpyHostToDevice);
  hipMemcpy(scene->d_triangles, scene->triangles, scene->triangleSize, hipMemcpyHostToDevice);
  hipMemcpy(scene->d_pointToObjIdxTable, scene->pointToObjIdxTable, scene->pointTableSize, hipMemcpyHostToDevice);
  hipMemcpy(scene->d_sceneobjects, scene->sceneobjects, scene->sceneObjectsSize, hipMemcpyHostToDevice);
  hipMemcpy(scene->d_transformMatrices, scene->transformMatrices, scene->matricesSize, hipMemcpyHostToDevice);
  hipMemcpy(d_scene, scene, sceneStructSize, hipMemcpyHostToDevice);
}

void cudaCleanup()
{
  hipHostFree(scene->transformMatrices);

  hipFree(scene->d_pointarray);
  hipFree(scene->d_trsfrmdpoints);
  hipFree(scene->d_triangles);
  hipFree(scene->d_sceneobjects);
  hipFree(scene->d_transformMatrices);
  hipFree(d_scene);
}

void rayTrace(
    uchar4 *pixelBuffer,
    const int bgColor)
{
  ZoneScopedN("rayTrace function");
  TracyCZoneN(cudaTrace, "Cuda trace", true);

  constexpr dim3 blockDim(RAYTRACE_BLOCK_SIDE, RAYTRACE_BLOCK_SIDE);
  constexpr dim3 gridDim((WIDTH + (RAYTRACE_BLOCK_SIDE - 1)) / RAYTRACE_BLOCK_SIDE,
                         (HEIGHT + (RAYTRACE_BLOCK_SIDE - 1)) / RAYTRACE_BLOCK_SIDE);

  float3_L f3lBg = intColToF3l(bgColor);

  rayTraceKernel<<<gridDim, blockDim>>>(
      pixelBuffer,
      d_scene,
      WIDTH, HEIGHT,
      f3lBg);

  TracyCZoneEnd(cudaTrace);
}