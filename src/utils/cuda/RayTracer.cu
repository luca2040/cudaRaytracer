#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdint>

#include "RayTracer.cuh"
#include "../DrawValues.h"
#include "../../math/cuda/CudaMath.cuh"

#include "kernels/RaytraceKernel.cuh"

#include "../../third_party/tracy/tracy/Tracy.hpp"

// ######## Device parameters ########

float3_L *d_pointarray;
triangleidx *d_triangles;

void cudaAllocateAndCopy(size_t pointsSize,
                         size_t triangleSize)
{
  // Allocate all the memory needed
  hipMalloc(&d_pointarray, pointsSize);
  hipMalloc(&d_triangles, triangleSize);

  // Copy the triangles index array, since its always static - not now because normals need to be recalculated each frame
  // hipMemcpy(d_triangles, triangles, triangleSize, hipMemcpyHostToDevice);
}

void cudaCleanup()
{
  hipFree(d_pointarray);
  hipFree(d_triangles);
}

void rayTrace(
    uchar4 *pixelBuffer,

    const float3_L camPos,
    const float3_L camViewOrigin,
    const float3_L imageX,
    const float3_L imageY,
    float inverseWidthMinus,
    float inverseHeightMinus,

    const float3_L *pointarray,
    const triangleidx *triangles,
    size_t triangleNum,

    size_t pointarraySize,
    size_t trianglesSize,

    const int bgColor)
{
  ZoneScopedN("rayTrace function");

  // hipMemcpy(d_pixelBuffer, pixelBuffer, pixelBufferSize, hipMemcpyHostToDevice);
  hipMemcpy(d_pointarray, pointarray, pointarraySize, hipMemcpyHostToDevice);
  hipMemcpy(d_triangles, triangles, trianglesSize, hipMemcpyHostToDevice);

  constexpr dim3 blockDim(16, 16);
  constexpr dim3 gridDim((WIDTH + 15) / 16, (HEIGHT + 15) / 16);

  rayTraceKernel<<<gridDim, blockDim>>>(
      pixelBuffer,
      camPos, camViewOrigin, imageX, imageY,
      inverseWidthMinus, inverseHeightMinus,
      d_pointarray, d_triangles, triangleNum,
      WIDTH, HEIGHT,
      bgColor);
}