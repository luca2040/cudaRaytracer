#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdint>

#include "RayTracer.cuh"
#include "../DrawValues.h"
#include "../../scene/structs/Scene.h"
#include "../../scene/structs/SceneObject.h"
#include "../../math/cuda/CudaMath.cuh"

#include "kernels/RaytraceKernel.cuh"

#include "../../third_party/tracy/tracy/Tracy.hpp"

// Device Scene
Scene *d_scene;
size_t sceneSize;

void cudaAllocateScene()
{
  // Allocate all the memory needed
  hipMalloc(&scene.d_pointarray, scene.pointsSize);
  hipMalloc(&scene.d_triangles, scene.triangleSize);
  hipMalloc(&scene.d_sceneobjects, scene.sceneObjectsSize);

  sceneSize = sizeof(Scene);
  hipMalloc(&d_scene, sceneSize);

  // Copy the triangles index array, since its always static - not anymore because normals need to be recalculated each frame
  // hipMemcpy(d_triangles, triangles, triangleSize, hipMemcpyHostToDevice);
}

void cudaCleanup()
{
  hipFree(scene.d_pointarray);
  hipFree(scene.d_triangles);
  hipFree(scene.d_sceneobjects);

  hipFree(d_scene);
}

void rayTrace(
    uchar4 *pixelBuffer,
    const int bgColor)
{
  ZoneScopedN("rayTrace function");

  // hipMemcpy(d_pixelBuffer, pixelBuffer, pixelBufferSize, hipMemcpyHostToDevice);
  hipMemcpy(scene.d_pointarray, scene.transformedPoints, scene.pointsSize, hipMemcpyHostToDevice);
  hipMemcpy(scene.d_triangles, scene.triangles, scene.triangleSize, hipMemcpyHostToDevice);
  hipMemcpy(scene.d_sceneobjects, scene.sceneobjects, scene.sceneObjectsSize, hipMemcpyHostToDevice);

  hipMemcpy(d_scene, &scene, sceneSize, hipMemcpyHostToDevice);

  constexpr dim3 blockDim(16, 16);
  constexpr dim3 gridDim((WIDTH + 15) / 16, (HEIGHT + 15) / 16);

  float3_L f3lBg = intColToF3l(bgColor);

  rayTraceKernel<<<gridDim, blockDim>>>(
      pixelBuffer,
      d_scene,
      WIDTH, HEIGHT,
      f3lBg);
}